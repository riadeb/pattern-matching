/**
 * APPROXIMATE PATTERN MATCHING
 *
 * INF560
 */
 #include <stdio.h>
 #include <string.h>
 #include <stdlib.h>
 #include <fcntl.h>
 #include <unistd.h>
 #include <sys/time.h>


 #include <hip/hip_runtime.h>
 

 #define APM_DEBUG 0

 #define MIN3(a, b, c) ((a) < (b) ? ((a) < (c) ? (a) : (c)) : ((b) < (c) ? (b) : (c)))
 #define MIN(a, b) ((a) < (b) ? (a) : (b) )

int * results_th ;

 void CHECK(hipError_t err){
    if( err != hipSuccess) {
        printf("Error ! %s \n", hipGetErrorString(err));
        exit(1);
    }  
 }

__global__ void compMatches(char* pattern,char * buf,int cuda_end ,int n_bytes, int size_pattern, int approx_factor, int * resultsth){
   
    int distance = 0 ;
     int size ;
     int i = threadIdx.x + blockIdx.x*blockDim.x;
     int n_th = gridDim.x*blockDim.x;
     resultsth[i] = 0;
     if(i == 0)     printf("cuda started \n");
       if(i < cuda_end){
        int * column = (int *)malloc( (size_pattern+1) * sizeof( int ) ) ;
           for(int j = i; j < cuda_end; j += n_th){
               size = size_pattern ;
               if ( n_bytes - j < size_pattern )
               {
                   size = n_bytes - j ;
               }
               int len = size;
               unsigned int x, y, lastdiag, olddiag;

               for (y = 1; y <= len; y++)
               {
                   column[y] = y;
               }
               for (x = 1; x <= len; x++) {
                   column[0] = x;
                   lastdiag = x-1 ;
                   for (y = 1; y <= len; y++) {
                       olddiag = column[y];
                       column[y] = MIN3(
                               column[y] + 1, 
                               column[y-1] + 1, 
                               lastdiag + (pattern[y-1] == buf[j+x-1] ? 0 : 1)
                               );
                       lastdiag = olddiag;

                   }
               }
               distance = column[len];
               if ( distance <= approx_factor ) {
                   resultsth[i]++ ;
               }
         }
       }
}
extern "C" void kernelCall(char* cpattern,char * cbuf,int cuda_end ,int n_bytes, int size_pattern, int approx_factor, int * results_th , int  nth_b,int nblock){

    compMatches<<<nblock, nth_b>>>(cpattern,cbuf,cuda_end,n_bytes,size_pattern,approx_factor,results_th);
    CHECK(hipGetLastError());
}
extern "C" int  finalcudaCall(char* cpattern,char * cbuf, int cuda_end, int * results_th , int  nth_b,int nblock){
    int * results;
    int nth = nth_b*nblock;
    results = (int *)malloc(nth* sizeof(int));
    CHECK(hipDeviceSynchronize());
   printf("cuda done\n");
    CHECK(hipMemcpy(results,results_th, nth* sizeof(int), hipMemcpyDeviceToHost));

    int res = 0;
    for(int j = 0; j < nth && j < cuda_end; j++){
        res += results[j];
    }
    return res;
}

 extern "C" char * cuda_malloc_cp(char *buf, int size){
    char *dBuf;
    CHECK(hipMalloc( (void**)&dBuf, size));
    CHECK(hipMemcpy(dBuf, buf, size, hipMemcpyHostToDevice) );
    return dBuf;
}
extern "C" int * cuda_malloc(int size){
    int *dBuf;
    CHECK(hipMalloc( (void **)&dBuf, size));
    return dBuf;
}